#include "hip/hip_runtime.h"
﻿//The polarization energy of the light is calculated and then multiplied by the Fresnel reflectance to get the final reflectivity:
//Ray:Ray normalization vector
//OBJ_N:The normal of the object
//Ray_N:The ray comes from the normal of the object
//PColor:P-polarized light carried by rays
//SColor:S-polarized light carried by rays

static __forceinline__ __device__ RpRs CalculatePolarization(float3& Ray, float3& OBJ_N, float3& Ray_N, float3& PColor, float3& SColor)
{
    float3 TangentA = Normalize(cross(OBJ_N, OBJ_N + float3{ 1.2349f,2.123f,-3.4857f }));
    float3 TangentB = Normalize(cross(TangentA, OBJ_N));
    float3 P_polarization = Normalize(cross(Ray, Ray_N));
    float3 S_polarization = Normalize(cross(Ray, P_polarization));

    float PN = dot(P_polarization, OBJ_N);
    float SN = dot(S_polarization, OBJ_N);
    float3 P = PN * PN * PColor + SN * SN * SColor;

    float PTA = dot(P_polarization, TangentA);
    float PTB = dot(P_polarization, TangentB);
    float STA = dot(S_polarization, TangentA);
    float STB = dot(S_polarization, TangentB);
    float3 S = PTA * PTA * PColor + STA * STA * SColor + PTB * PTB * PColor + STB * STB * SColor;

    RpRs temp;
    temp.Rp = P;
    temp.Rs = S;
    return temp;
}




